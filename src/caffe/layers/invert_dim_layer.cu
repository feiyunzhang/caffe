#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/invert_dim_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void invert_dim(const int count, const int invert_dim,
    const int outer_dim, const int inner_dim,
    const Dtype* input, Dtype* output) {
  CUDA_KERNEL_LOOP(index, count) {

    int n = index % inner_dim;
    int i = index / inner_dim % invert_dim;
    int o = index / outer_dim;
    //int output_index = o * outer_dim + (invert_dim - 1 - i) * inner_dim + n;
    //printf("Index %d, o %d, i %d, n %d, output index %d\n", index, o, i, n, output_index);

    output[o * outer_dim + (invert_dim - 1- i) * inner_dim + n] = input[index];
  }
}

template <typename Dtype>
void InvertDimLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = bottom[0]->count();

  //printf("count %d, invert_dim %d, outer_dim %d, inner_dim %d\n", count, invert_dim_, outer_dim_, inner_dim_);
  invert_dim<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, invert_dim_, outer_dim_, inner_dim_, bottom_data, top_data);
}

template <typename Dtype>
void InvertDimLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();
  int count = bottom[0]->count();

  invert_dim<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>
    (count, invert_dim_, outer_dim_, inner_dim_, top_diff, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(InvertDimLayer);


}  // namespace caffe
